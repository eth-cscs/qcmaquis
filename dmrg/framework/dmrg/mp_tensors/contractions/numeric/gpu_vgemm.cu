#include "hip/hip_runtime.h"
/*****************************************************************************
 *
 * ALPS MPS DMRG Project
 *
 * Copyright (C) 2017 Department of Chemistry and the PULSE Institute, Stanford University
 *                    Laboratory for Physical Chemistry, ETH Zurich
 *               2017-2017 by Sebastian Keller <sebkelle@phys.ethz.ch>
 * 
 * This software is part of the ALPS Applications, published under the ALPS
 * Application License; you can use, redistribute it and/or modify it under
 * the terms of the license, either version 1 or (at your option) any later
 * version.
 * 
 * You should have received a copy of the ALPS Application License along with
 * the ALPS Applications; see the file LICENSE.txt. If not, the license is also
 * available from http://alps.comp-phys.org/.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR 
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
 * FITNESS FOR A PARTICULAR PURPOSE, TITLE AND NON-INFRINGEMENT. IN NO EVENT 
 * SHALL THE COPYRIGHT HOLDERS OR ANYONE DISTRIBUTING THE SOFTWARE BE LIABLE 
 * FOR ANY DAMAGES OR OTHER LIABILITY, WHETHER IN CONTRACT, TORT OR OTHERWISE, 
 * ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
 * DEALINGS IN THE SOFTWARE.
 *
 *****************************************************************************/

//#include <new>
#include <cassert>
//#include <complex>
#include <cstddef>
#include <cstdlib>
#include <limits>
#include <cstring>
#include <malloc.h>
#include <stdint.h>
#include <iostream>
//#include <boost/static_assert.hpp>
// BLAS declarations
//#include <boost/numeric/bindings/blas/detail/blas.h>

#include "gpu.h"
//#include "common.h"


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void accumulate(float *in, float *out, size_t N, size_t chunks)
{
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    float temp = 0;
    while (tid < N) {
        for (size_t i = 0; i < chunks; ++i)
            temp += in[tid + N*i] ;

        out[tid] = temp;
        tid += blockDim.x * gridDim.x;
    }
}

template <class T>
__global__ void set_batch(T** batch, T* a, T* dev_t, size_t t_size, size_t N)
{
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < N) {

        batch[tid] = a;
        batch[2*N + tid] = dev_t + tid * t_size;

        tid += blockDim.x * gridDim.x;
    }
}

template <class T>
void batched_gemm_tpl(hipblasHandle_t handle, BatchGemmData<T> & batch, int M, int N, size_t t_size, T* mpsdata, T* dev_t)
{
    hipblasOperation_t cuop[2] = {HIPBLAS_OP_N, HIPBLAS_OP_T};
    T one = 1.0;
    T zero = 0.0;

    set_batch<<<1, 64>>>(batch.dev_b, mpsdata + batch.in_offset * M, dev_t + batch.tstart * t_size, t_size, batch.size);

    hipblasDgemmBatched(handle, cuop[0], cuop[batch.trans], M, N, batch.K, &one,
                       (const T**)(batch.dev_b), M,
                       (const T**)(batch.dev_b + batch.size), batch.LDB, &zero,
                       batch.dev_b + 2*batch.size, M, batch.size
                       );
}

#define TILE_DIM 32
#define BLOCK_ROWS 8

template <class T>
__global__ void cuda_copy_v(unsigned N, unsigned M, unsigned cnt, T** dev_a, T* dev_tra)
{
    unsigned x = threadIdx.x + blockIdx.x * TILE_DIM;
    unsigned y = threadIdx.y + blockIdx.y * TILE_DIM;

    size_t mz = blockIdx.z;
    while (mz < cnt)
    {
        size_t out = mz * N * M; 
        for (unsigned my = y; my < M + TILE_DIM; my += gridDim.y * TILE_DIM)
        {
            for (unsigned mx = x; mx < N + TILE_DIM; mx += gridDim.x * TILE_DIM)
            {
                #pragma unroll
                for (unsigned j = 0; j < TILE_DIM; j+=BLOCK_ROWS)
                {
                    size_t offset = mx + (my+j) * N;
                    if (mx < N && (my+j) < M)
                       dev_tra[out + offset] = dev_a[mz][offset];
                }
            }
        }
        mz += gridDim.z;
    }
}

template <class T>
__global__ void cuda_transpose_v(unsigned N, unsigned M, unsigned cnt, T** dev_a, T* dev_tra)
{
    __shared__ T tile[TILE_DIM][TILE_DIM+1];

    unsigned x = threadIdx.x + blockIdx.x * TILE_DIM;
    unsigned y = threadIdx.y + blockIdx.y * TILE_DIM;

    ///size_t i = blockIdx.z;
    //size_t mz = blockIdx.z;
    //while (mz < cnt)
    //{
        //size_t out = mz * N * M; 
        for (unsigned my = y; my < M + TILE_DIM; my += gridDim.y * TILE_DIM)
        {
            for (unsigned mx = x; mx < N + TILE_DIM; mx += gridDim.x * TILE_DIM)
            {
                for (unsigned mz = blockIdx.z; mz < cnt; mz += gridDim.z)
                {
                    size_t out = mz * N * M;
                    #pragma unroll
                    for (unsigned j = 0; j < TILE_DIM; j+=BLOCK_ROWS)
                    {
                        size_t offset = mx + (my+j) * N;
                        if (mx < N && (my+j) < M)
                        {
                           tile[threadIdx.y+j][threadIdx.x] = dev_a[mz][offset];
                        }
                    }

                    __syncthreads();

                    #pragma unroll
                    for (unsigned j = 0; j < TILE_DIM; j+=BLOCK_ROWS)
                    {
                        unsigned tx = my-threadIdx.y + threadIdx.x;
                        unsigned ty = mx-threadIdx.x + threadIdx.y + j;
                        size_t tr_offset = tx + ty * M;
                        if (tx < M && ty < N)
                           dev_tra[out + tr_offset] = tile[threadIdx.x][threadIdx.y+j];
                    }

                    __syncthreads();
                }
            }
        }
        //mz += gridDim.z;
    //}
}


template <class T>
void coalesced_gemm_tpl(hipblasHandle_t handle, BatchGemmData<T> & batch, int M, int N, size_t t_size, T* mpsdata, T* dev_t, T* r_buf)
{
    hipblasOperation_t cuop[2] = {HIPBLAS_OP_N, HIPBLAS_OP_T};
    T one = 1.0;
    T zero = 0.0;

    size_t b_size = batch.K * N;
    dim3 threads(TILE_DIM, BLOCK_ROWS);
    dim3 blocks3d(2,2,std::min(batch.size, 65535lu));

    if (batch.trans)
        //for (size_t k = 0; k < batch.b.size(); ++k)
        //    hipblasDgeam(handle, cuop[1], cuop[0], batch.K, N,
        //                &one, batch.b[k], batch.LDB,
        //                &zero, batch.b[k], batch.K,
        //                r_buf + k*b_size, batch.K);
        cuda_transpose_v<<<blocks3d, threads>>>(N, batch.K, batch.size, batch.dev_b + batch.size, r_buf);
    else
        //for (size_t k = 0; k < batch.b.size(); ++k)
        //    hipMemcpy( r_buf + k * b_size, batch.b[k], b_size* sizeof(T), hipMemcpyDeviceToDevice);
        cuda_copy_v<<<blocks3d, threads>>>(batch.K, N, batch.size, batch.dev_b + batch.size, r_buf);


    hipblasDgemm(handle, cuop[0], cuop[0], M, N * batch.b.size(), batch.K, &one,
                mpsdata + batch.in_offset * M, M,
                r_buf, batch.K, &zero, dev_t + batch.tstart * t_size, M);
}


void vgemm(hipblasHandle_t handle, BatchGemmData<double> & batch, int M, int N, size_t t_size, double* mpsdata, double* dev_t, double* r_buf)
{
   coalesced_gemm_tpl(handle, batch, M, N, t_size, mpsdata, dev_t, r_buf);
   //batched_gemm_tpl(handle, batch, M, N, t_size, mpsdata, dev_t);
}
